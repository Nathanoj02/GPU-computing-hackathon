// #define ENABLE_NVTX
// #define ENABLE_CPU_BASELINE
// #define DEBUG_PRINTS
#define ENABLE_CORRECTNESS_CHECK

#define EXIT_INCORRECT_DISTANCES 10

#include <stdio.h>
#include <hip/hip_runtime.h>

#ifdef ENABLE_NVTX
#include <nvtx3/nvToolsExt.h>
#endif

#include "../distributed_mmio/include/mmio.h"
#include "../distributed_mmio/include/mmio_utils.h"

#include "../include/colors.h"
#include "../include/utils.cuh"
#include "../include/cli.hpp"
#include "../include/mt19937-64.hpp"
#include "../include/bfs_baseline.cuh"

void gpu_bfs(
  const uint32_t N,           // Number of veritices
  const uint32_t M,           // Number of edges
  const uint32_t *h_rowptr,   // Graph CSR rowptr
  const uint32_t *h_colidx,   // Graph CSR colidx
  const uint32_t source,      // Source veritex
  int *h_distances            // Write here your distances
) {
  /***********************
   * IMPLEMENT HERE YOUR CUDA BFS
   * Feel free to structure you code (i.e. create other files, macros etc.)
   * *********************/

  // !! This is just a placeholder !!
  gpu_bfs_baseline(N, M, h_rowptr, h_colidx, source, h_distances, true);

  // !! This is an example of how to keep track of runtime. Make sure to include everything. !!
  /* float tot_time = 0.0f;
  CPU_TIMER_INIT(BFS_preprocess)

  <<< preprocess >>>

  CHECK_CUDA(hipDeviceSynchronize());
  CPU_TIMER_STOP(BFS_preprocess)
  tot_time += CPU_TIMER_ELAPSED(BFS_preprocess);
  CPU_TIMER_PRINT(BFS_preprocess)

  CPU_TIMER_INIT(BFS)

  <<< kernel >>>

  CHECK_CUDA(hipDeviceSynchronize());
  CPU_TIMER_STOP(BFS)
  tot_time += CPU_TIMER_ELAPSED(BFS);
  CPU_TIMER_PRINT(BFS)

  CPU_TIMER_INIT(BFS_postprocess)

  <<< postprocess >>>

  CHECK_CUDA(hipDeviceSynchronize());
  CPU_TIMER_STOP(BFS_postprocess)
  tot_time += CPU_TIMER_ELAPSED(BFS_postprocess);
  CPU_TIMER_PRINT(BFS_postprocess)

  // This output format is MANDATORY, DO NOT CHANGE IT
  printf("\n[OUT] Total BFS time: %f ms\n" RESET, tot_time); */
}

int main(int argc, char **argv) {
  int return_code = EXIT_SUCCESS;

  Cli_Args args;
  init_cli();
  if (parse_args(argc, argv, &args) != 0) {
    return -1;
  }

  int device_count;
  hipGetDeviceCount(&device_count);
  if (device_count <= 0) {
    fprintf(stderr, "No GPU available: device_count=%d\n", device_count);
    return EXIT_FAILURE;
  }
  hipSetDevice(0);

  CPU_TIMER_INIT(MTX_read)
  CSR_local<uint32_t, float> *csr = Distr_MMIO_CSR_local_read<uint32_t, float>(args.filename);
  if (csr == NULL) {
    printf("Failed to import graph from file [%s]\n", args.filename);
    return -1;
  }
  CPU_TIMER_STOP(MTX_read)
  printf("\n[OUT] MTX file read time: %f ms\n", CPU_TIMER_ELAPSED(MTX_read));
  printf("Graph size: %.3fM vertices, %.3fM edges\n", csr->nrows / 1e6, csr->nnz / 1e6);

  GraphCSR graph;
  graph.row_ptr = csr->row_ptr;
  graph.col_idx = csr->col_idx;
  graph.num_vertices = csr->nrows;
  graph.num_edges = csr->nnz;
  // print_graph_csr(graph);

  uint32_t *sources = generate_sources(&graph, args.runs, graph.num_vertices, args.source);
  int *distances_gpu_baseline = (int *)malloc(graph.num_vertices * sizeof(int));
  int *distances = (int *)malloc(graph.num_vertices * sizeof(int));
  bool correct = true;

  for (int source_i = 0; source_i < args.runs; source_i++) {
    uint32_t source = sources[source_i];
    printf("\n[OUT] -- BFS iteration #%u, source=%u --\n", source_i, source);

    // Run the BFS baseline
    gpu_bfs_baseline(graph.num_vertices, graph.num_edges, graph.row_ptr, graph.col_idx, source, distances_gpu_baseline, false);

    #ifdef ENABLE_NVTX
		  nvtxRangePushA("Complete BFS");
    #endif
    gpu_bfs(graph.num_vertices, graph.num_edges, graph.row_ptr, graph.col_idx, source, distances);
    #ifdef ENABLE_NVTX
		  nvtxRangePop();
    #endif

    bool match = true;
    #ifdef ENABLE_CORRECTNESS_CHECK
      for (uint32_t i = 0; i < graph.num_vertices; ++i) {
        if (distances_gpu_baseline[i] != distances[i]) {
          printf("Mismatch at node %u: Baseline distance = %d, Your distance = %d\n", i, distances_gpu_baseline[i], distances[i]);
          match = false;
          break;
        }
      }
      if (match) {
        printf(BRIGHT_GREEN "Correctness OK\n" RESET);
      } else {
        printf(BRIGHT_RED "GPU and CPU BFS results do not match for source node %u.\n" RESET, source);
        return_code = EXIT_INCORRECT_DISTANCES;
        correct = false;
      }
    #endif

    #ifdef ENABLE_CPU_BASELINE
      int cpu_distances[graph.num_vertices];

      CPU_TIMER_INIT(CPU_BFS)
      cpu_bfs_baseline(graph.num_vertices, graph.row_ptr, graph.col_idx, source, cpu_distances);
      CPU_TIMER_CLOSE(CPU_BFS)

      match = true;
      for (uint32_t i = 0; i < graph.num_vertices; ++i) {
        if (distances_gpu_baseline[i] != cpu_distances[i]) {
          printf("Mismatch at node %u: GPU distance = %d, CPU distance = %d\n", i, distances_gpu_baseline[i], cpu_distances[i]);
          match = false;
          break;
        }
      }
      if (match) {
        printf(BRIGHT_GREEN "[CPU] Correctness OK\n" RESET);
      } else {
        printf(BRIGHT_RED "GPU and CPU BFS results do not match for source node %u.\n" RESET, source);
        return_code = EXIT_INCORRECT_DISTANCES;
      }
    #endif
  }

  if (correct) printf("\n[OUT] ALL RESULTS ARE CORRECT\n");
  else         printf(BRIGHT_RED "\nSOME RESULTS ARE WRONG\n" RESET);

  Distr_MMIO_CSR_local_destroy(&csr);
  free(sources);
  free(distances_gpu_baseline);
  free(distances);

  return return_code;
}
